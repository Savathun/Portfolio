#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define Max_2d_Block_Dim 32
#define Max_Concurent_Blocks 56
#define Max_Block_Dim 1024
#define HANDLE_ERROR( err )  ( HandleError( err, __FILE__, __LINE__ ) )
// I made this using the stats_gpu_template.cu as a guide 
typedef struct {
    unsigned int index;
    double value;
} local_max;

void HandleError(hipError_t err, const char* file, int line) {
    //
    // Handle and report on CUDA errors.
    //
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);

        exit(EXIT_FAILURE);
    }
}

void checkCUDAError(const char* msg, bool exitOnError) {
    //
    // Check cuda error and print result if appropriate.
    //
    hipError_t err = hipGetLastError();

    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        if (exitOnError) {
            exit(-37);
        }
    }
}

void cleanupCuda(void) {
    //
    // Clean up CUDA resources.
    //

    //
    // Explicitly cleans up all runtime-related resources associated with the
    // calling host thread.
    //
    HANDLE_ERROR(
        hipDeviceReset()
    );
}

void columnMaxKernelDim(unsigned int& blocks, unsigned int& threads, const unsigned int& size) {
    blocks = (size / Max_Block_Dim) + ((size % Max_Block_Dim) ? 1 : 0);
    threads = Max_Block_Dim;
    // blocks are the numeber of blocks when spliting by Max_Block_Dim. It will add another if necessary 
    // threads are equal to Max_Block_Dim by default
     
    if (size < threads) {
        blocks = 1; // makes clear that only 1 block is desired 
        if (size < 2) threads = 2;
        else if (size < 4) threads = 4;
        else if (size < 8) threads = 8;
        else if (size < 16) threads = 16;
        else if (size < 32) threads = 32;
        else if (size < 64) threads = 64;
        else if (size < 128) threads = 128;
        else if (size < 256) threads = 256;
        else if (size < 512) threads = 512;
        else if (size < 1024) threads = 1024;
    }
    // if there are less elements than there are allotted in single block will limit the threads allocated in the gpu
}

void swapRowsKernelDim(unsigned int& blocks, unsigned int& threads, const unsigned int& size) {
    blocks = (size / Max_Block_Dim) + ((size % Max_Block_Dim) ? 1 : 0);
    threads = Max_Block_Dim;

    if (size < threads){
        threads = size;
    }
}

void gaussianReductionKernelDim(unsigned int& blocks, unsigned int& threads, const unsigned int& size) {
    blocks = (size / Max_2d_Block_Dim) + ((size % Max_2d_Block_Dim) ? 1 : 0);
    threads = Max_2d_Block_Dim;

    if (size < threads){ 
        threads = size; 
    }
}

// ---------------------------------------------------------------------------------------------------------------------
// GPU KERNELS

__device__ double device_abs(double x) {
    //
    // Calculate x^y on the GPU.
    //
    return fabs(x);
}

__device__ double device_invert_sign(double x) {
    // returns the opposite sign of x
    if (x < 0.0) return 1.0;
    else return -1.0;
}

__global__ void columnMax(double* A, unsigned int* size, unsigned int* incolumn, local_max* outdata) {
    extern __shared__ double sdata[];           // container for the data being processed 
    __shared__ unsigned int index_partition;    // beginning for the index partition 
    __shared__ unsigned int N;                  // size of matrix (NxN)
    __shared__ unsigned int column;             // columns we find the max in

    unsigned int tid = threadIdx.x;

    if (tid == 0) {
        index_partition = blockDim.x;
        N = *size;
        column = *incolumn;
    }

    __syncthreads();

    int row = blockDim.x * blockIdx.x + tid;      // the abs position for within the matrix
    if (row >= column && row < N) {
        sdata[tid] = A[row * N + column];     // puts the matrix value into sdata
        sdata[tid + index_partition] = row;        // puts the matrix index into sdata
    }
    else {
        sdata[tid] = 0;     // puts the matrix value into sdata
        sdata[tid + index_partition] = -1;        // puts the matrix index into sdata
    }

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            double absL = device_abs(sdata[tid]);   // abs value of the left 
            double absR = device_abs(sdata[tid + s]);   // abs value of the right
            if (absL < absR) {
                // if the right num is larger
                sdata[tid] = sdata[tid + s];
                sdata[tid + index_partition] = sdata[tid + s + index_partition];
                // this empties  the right half of the array but is fine because it isn't needed anymore
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        outdata[blockIdx.x].index = sdata[index_partition];
        outdata[blockIdx.x].value = sdata[0];
    }
}

__global__ void swapRows(double* A, unsigned int* size, unsigned int* top_row, unsigned int* bot_row)
{
    extern __shared__ double sdata[];           // container for the data processing
    __shared__ unsigned int N;                  // size of matrix (NxN)
    __shared__ unsigned int top;                // the top row that is swapped with the bottom row
    __shared__ unsigned int bot;                // the bottom row that is swapped with the top row

    unsigned int tid = threadIdx.x;

    if (tid == 0) {
        N = *size; 
        top = *top_row;
        bot = *bot_row;
    }

    __syncthreads();

    // abs position within the matrix
    int column = blockDim.x * blockIdx.x + tid;

    // only swap the sub matrix A[row][row] because everything above and to the left of row is dealt with
    // above row data in U
    // left of row data in L

    // load the top row into shared memory
    if (column < N) {
        sdata[tid] = A[top * N + column];
        A[top * N + column] = A[bot * N + column];
        A[bot * N + column] = sdata[tid];
    }
}

__global__ void findCoefficients(double* A, unsigned int* size, unsigned int* inboundary) {
    extern __shared__ double sdata[];           // container for processing data
    __shared__ unsigned int N;                  // size of matrix (NxN)
    __shared__ unsigned int boundary;           // the top row that is swapped with the bottom row
    __shared__ double denominator;              // the number to go into the denominator
    double coefficient;

    unsigned int tid = threadIdx.x;

    if (tid == 0) {
        boundary = *inboundary;
        N = *size;
        denominator = A[boundary * N + boundary]; // top left value
    }

    __syncthreads();

    unsigned int row = blockDim.x * blockIdx.x + tid;
    if (row > boundary && row < N) {
        sdata[tid] = A[row * N + boundary];
        coefficient = sdata[tid] / denominator;
        A[row * N + boundary] = coefficient;
    }
}

__global__ void gaussianReduction(double* A, unsigned int* size, unsigned int* inboundary) {
    __shared__ unsigned int N;                  // size of matrix (NxN)
    __shared__ unsigned int boundary;           // the top row that is swapped with the bottom row
    double coefficient;                         // the number for multiplication of the top row (the number in L)
    double R_bot;                               // the element in the bottom row
    double R_top;                               // the number in the top row multipied by coefficient


    unsigned int tidx = threadIdx.x;
    unsigned int tidy = threadIdx.y;
    unsigned int absolute_row = blockDim.x * blockIdx.x + tidx;
    unsigned int absolute_column = blockDim.y * blockIdx.y + tidy;

    if (tidx == 0 && tidy == 0) {
        N = *size;
        boundary = *inboundary;
    }

    __syncthreads();

    if ((absolute_column > boundary && absolute_column < N) && (absolute_row > boundary && absolute_row < N)) {
        coefficient = A[absolute_row * N + boundary];
        coefficient *= device_invert_sign(A[absolute_row * N + absolute_column]);
        R_bot = A[absolute_row * N + absolute_column];
        R_top = A[boundary * N + absolute_column];
        A[absolute_row * N + absolute_column] = R_bot + coefficient * R_top;
    }
}
// ---------------------------------------------------------------------------------------------------------------------

int main(int argc, char* argv[]) {
    unsigned int seed;
    struct timeval start, end;
    double runtime;
    bool singular = false;
    bool toPrint = false;

    if (argc < 2) {
        printf("Format: stats_gpu <size of array> <random seed> <print>\n");
        printf("Arguments:\n");
        printf("  size of array - The size of the matrix to be generated and then processed\n");
        printf("                  generator that will generate the contents of the array\n");
        printf("                  to be processed\n");

        exit(1);
    }

    // Get the size of matrix 
    unsigned int array_size = atoi(argv[1]);
    unsigned int array_area = array_size * array_size;

    // Get the print variable
    if (argv[3])
    {
        if (array_size < 10)
        {
            toPrint = atoi(argv[3]);
        }
        else
        {
            toPrint = 0;
        }
    }
    else
    {
        toPrint = 0;
    }


    // Get the seed to be used
    seed = atoi(argv[2]);

    // Make sure that CUDA resources get cleaned up on exit.
    atexit(cleanupCuda);

    // Record the start time.
    gettimeofday(&start, NULL);

    // Allocate the array to be populated
    double* array = (double*)malloc(array_area * sizeof(double));
    printf("Array size: %d\nArray Area: %d\nSeed: %d\n", array_size, array_area, seed);

    // initialize the 2d array/ matrix
    srand(seed);
    for (int i = 0; i < array_size; i++) {
        for (int j = 0; j < array_size; ++j) {
            //printf("[%d]\n", i*array_size + j);
            array[i * array_size + j] = ((double)rand() / (double)RAND_MAX);
        }
    }

    if (toPrint == true)
    {
        printf("Matrix A\n");
        for (int i = 0; i < array_size; i++) {
            printf("Row %d: ", i);
            for (int j = 0; j < array_size; ++j) {
                printf("%.2f\t", array[i * array_size + j]);
            }
            printf("\n");
        }
    }

    // write the matrix address to GPU
    double* A; // for copying array to the GPU
    HANDLE_ERROR(hipMalloc((void**)&A, array_area * sizeof(double)));
    HANDLE_ERROR(hipMemcpy(A, array, array_area * sizeof(double), hipMemcpyHostToDevice));

    // write the array size to GPU
    unsigned int* array_size_ptr;
    HANDLE_ERROR(hipMalloc((void**)&array_size_ptr, sizeof(unsigned int)));
    HANDLE_ERROR(hipMemcpy(array_size_ptr, &array_size, sizeof(unsigned int), hipMemcpyHostToDevice));

    unsigned int* row_to_swap_ptr;
    HANDLE_ERROR(hipMalloc((void**)&row_to_swap_ptr, sizeof(unsigned int)));

    unsigned int* current_row_ptr;
    HANDLE_ERROR(hipMalloc((void**)&current_row_ptr, sizeof(unsigned int)));


    // Record the end time.
    gettimeofday(&end, NULL);

    // -----------------------------------------------------------------------------------------------------------------
    // LU Decomp
    // -----------------------------------------------------------------------------------------------------------------

    // kernel variables
    unsigned int numthreads, numblocks, sdata_size; // the number of threads and blocks assigned 


    columnMaxKernelDim(numblocks, numthreads, array_size);
    local_max* max_data = new local_max[numblocks];
    local_max* max_data_ptr;
    HANDLE_ERROR(hipMalloc((void**)&max_data_ptr, numblocks * sizeof(local_max)));

    for (unsigned int i = 0; i < array_size - 1; ++i) {
        // copies column being used to gpu
        HANDLE_ERROR(hipMemcpy(current_row_ptr, &i, sizeof(unsigned int), hipMemcpyHostToDevice));

        // find the max in the column for the pivot
        columnMaxKernelDim(numblocks, numthreads, array_size);
        sdata_size = numthreads * 2;
        
        //        printf("SData Size: %d\n", sdata_size);
        //        printf("Numblocks: %d\n", numblocks);
        //        printf("Numthreads: %d\n", numthreads);
        //used in testing 
        
        //calls kernal finds the max
        columnMax << < numblocks, numthreads, sdata_size * sizeof(double) >> > (A, array_size_ptr, current_row_ptr, max_data_ptr);

        // allows the gpu to finish
        hipDeviceSynchronize();

        HANDLE_ERROR(hipMemcpy(max_data, max_data_ptr, numblocks * sizeof(local_max), hipMemcpyDeviceToHost));

        unsigned int max_column_index = max_data[0].index;
        double max_column_value = max_data[0].value;
        for (int j = 0; j < numblocks; ++j) {
            if (fabs(max_data[j].value) > fabs(max_column_value) && max_data[j].index != -1.0) {
                max_column_index = max_data[j].index;
                max_column_value = max_data[j].value;
            }
        }

        //printf("Max row in column %d: %d\n", i, max_column_index);
        //testing finds to see if column max is right
        // singular matrix check
        if (max_column_value == 0.0) {
            printf("The matrix is singular!\n");
            singular = true;
            break;
        }

        HANDLE_ERROR(hipMemcpy(row_to_swap_ptr, &max_column_index, sizeof(unsigned int), hipMemcpyHostToDevice));
        swapRowsKernelDim(numblocks, numthreads, array_size);
        swapRows << < numblocks, numthreads, numthreads * sizeof(double) >> > (A, array_size_ptr, current_row_ptr, row_to_swap_ptr);

        // lets gpu finish
        hipDeviceSynchronize();

        swapRowsKernelDim(numblocks, numthreads, array_size);
        findCoefficients << < numblocks, numthreads, numthreads * sizeof(double) >> > (A, array_size_ptr, current_row_ptr);

        // lets gpu finish
        hipDeviceSynchronize();

        gaussianReductionKernelDim(numblocks, numthreads, array_size);
        dim3 grid(numblocks, numblocks);
        dim3 block(numthreads, numthreads);

        gaussianReduction << < grid, block >> > (A, array_size_ptr, current_row_ptr);

        // let gpu finish
        hipDeviceSynchronize();


        HANDLE_ERROR(hipMemcpy(array, A, array_area * sizeof(double), hipMemcpyDeviceToHost));
    }

    // -----------------------------------------------------------------------------------------------------------------
    // End LU Decomp
    // -----------------------------------------------------------------------------------------------------------------

    if (!singular) {
        HANDLE_ERROR(hipMemcpy(array, A, array_area * sizeof(double), hipMemcpyDeviceToHost));

        if (toPrint == true)
        {
            printf("\nMatrix L\n");
            for (int i = 0; i < array_size; i++) {
                printf("Row %d: ", i);
                for (int j = 0; j < array_size; ++j) {
                    if (j < i) {
                        printf("%.2f\t", array[i * array_size + j]);
                    }
                    else if (j == i) {
                        printf("%.2f\t", 1.0);
                    }
                    else {
                        printf("%.2f\t", 0.0);
                    }
                }
                printf("\n");
            }

            printf("\nMatrix U\n");
            for (int i = 0; i < array_size; i++) {
                printf("Row %d: ", i);
                for (int j = 0; j < array_size; ++j) {
                    if (j >= i) {
                        printf("%.2f\t", array[i * array_size + j]);
                    }
                    else {
                        printf("%.2f\t", 0.0);
                    }
                }
                printf("\n");
            }
        }
    }


    // Calculate the runtime.
    runtime = ((end.tv_sec - start.tv_sec) * 1000.0) + ((end.tv_usec - start.tv_usec) / 1000.0);
    printf("Processing Time: %4.4f milliseconds\n", runtime);

    // free cuda memory
    hipFree(array_size_ptr);
    hipFree(current_row_ptr);
    hipFree(A);
    hipFree(row_to_swap_ptr);
    hipFree(max_data_ptr);

    // Free the allocated array.
    //free( array );

    return 0;
}
